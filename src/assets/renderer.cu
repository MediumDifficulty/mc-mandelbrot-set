
#include <hip/hip_runtime.h>
extern "C" __global__ void render_kernel(char *out, int width, int height, int iternations, char *cached_values, int numel) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < numel) {
        int x = i % width;
        int y = i / width;

        double uv_x = ((double)x / (double)width) * 4.0 - 2.0;
        double uv_y = -(((double)y / (double)width) * 4.0 - 2.0);

        double cycle_x = uv_x;
        double cycle_y = uv_y;

        double cycle_xs = cycle_x*cycle_x;
        double cycle_ys = cycle_y*cycle_y;

        int pixel_iterations = 0;

        while (pixel_iterations < iternations && (cycle_xs + cycle_ys < 4.0)) {
            cycle_y = 2.0 * cycle_x*cycle_y + uv_y;
            cycle_x = cycle_xs - cycle_ys + uv_x;

            cycle_xs = cycle_x*cycle_x;
            cycle_ys = cycle_y*cycle_y;

            pixel_iterations++;
        }

        int colour;

        int colours[] = {
            0xEB1515,
            0xFF8000,
            0xFFFF00,
            0x00CC00,
            0x0080FF,
            0x4C0099,
            0x990099,
        };

        if (pixel_iterations >= iternations) {
            colour = 0;
        } else {
            colour = colours[pixel_iterations % 7];
        }

        out[i] = cached_values[colour];
    }
}